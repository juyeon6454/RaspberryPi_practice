#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>                     /* USHRT_MAX 상수를 위해서 사용한다. */
//#include <unistd.h>

#include "bmpHeader.h"

/* 이미지 데이터의 경계 검사를 위한 매크로 */
#define LIMIT_UBYTE(n) ((n)>UCHAR_MAX)?UCHAR_MAX:((n)<0)?0:(n)
#define widthbytes(bits) (((bits)+31)/32*4)

int x, y, z;
int elemSize;

typedef unsigned char ubyte;
//Cuda kernel for converting RGB image into a GreyScale image
__global__ void convertBlur(ubyte* in, ubyte* out, int rows, int cols, int elemSize) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    //int z = threadIdx.z + blockIdx.z * blockDim.z;

// define the kernel
    float kernel[3][3] = { {1 / 9.0, 1 / 9.0, 1 / 9.0},
                           {1 / 9.0, 1 / 9.0, 1 / 9.0},
                           {1 / 9.0, 1 / 9.0, 1 / 9.0} };

    int arr[9] = { 0 };

    for (z = 0; z < elemSize; z++) {

        for (y = 1; y < bmpInfoHeader.biHeight - 1; y++) {
            for (x = 1; x < (bmpInfoHeader.biWidth - 1) * elemSize; x += elemSize) {
                if ((x > 0) && (x < bmpInfoHeader.biWidth * elemSize) && (y > 0)
                    && (y < bmpInfoHeader.biHeight)) {

                    float sum = 0.0;
                    for (int i = -1; i < 2; i++) {
                        for (int j = -1; j < 2; j++) {
                            sum += kernel[i + 1][j + 1] * inimg[(x - i * elemSize) + (y - j) * size + z];
                        }
                    }
                    outimg[x + y * size + z] = sum;
                }
            }
        }


        for (x = 1; x < (bmpInfoHeader.biWidth - 2) * elemSize; x++) {
            float sum = 0.0;
            arr[0] = arr[3] = inimg[(x - 1) * elemSize + z] * kernel[0][1];
            arr[1] = arr[4] = inimg[x * elemSize + z] * kernel[1][1];
            arr[2] = arr[5] = inimg[(x + 1) * elemSize + z] * kernel[2][1];
            arr[6] = inimg[(x - 1) * elemSize + size + z] * kernel[0][2];
            arr[7] = inimg[x * elemSize + size + z] * kernel[1][2];
            arr[8] = inimg[(x + 1) * elemSize + size + z] * kernel[2][2];

            sum = arr[0] + arr[1] + arr[2] + arr[3] + arr[4] + arr[5] + arr[6]
                + arr[7] + arr[8];
            outimg[x * elemSize + z] = sum;
        }

        for (x = 1; x < (bmpInfoHeader.biWidth - 2) * elemSize; x++) {
            float sum = 0.0;
            arr[6] = arr[3] = inimg[(x - 1) * elemSize +
                size * (bmpInfoHeader.biHeight - 1) + z] * kernel[0][1];

            arr[7] = arr[4] = inimg[x * elemSize +
                size * (bmpInfoHeader.biHeight - 1) + z] * kernel[1][1];

            arr[8] = arr[5] = inimg[(x + 1) * elemSize +
                size * (bmpInfoHeader.biHeight - 1) + z] * kernel[2][1];

            arr[0] = inimg[(x - 1) * elemSize +
                size * (bmpInfoHeader.biHeight - 2) + z] * kernel[0][0];

            arr[1] = inimg[x * elemSize +
                size * (bmpInfoHeader.biHeight - 2) + z] * kernel[1][0];

            arr[2] = inimg[(x + 1) * elemSize +
                size * (bmpInfoHeader.biHeight - 2) + z] * kernel[2][0];

            sum = arr[0] + arr[1] + arr[
                2] + arr[3] + arr[4] + arr[5] + arr[6]
                    + arr[7] + arr[8];
                outimg[x * elemSize +
                    size * (bmpInfoHeader.biHeight - 1) + z] = sum;

        }

        for (y = 1; y < bmpInfoHeader.biHeight - 1; y++) {
            float sum = 0.0;
            arr[0] = arr[1] = inimg[(y - 1) * size + z] * kernel[1][0];
            arr[3] = arr[4] = inimg[(y)*size + z] * kernel[1][1];
            arr[6] = arr[7] = inimg[(y + 1) * size] * kernel[1][2];
            arr[2] = inimg[elemSize + (y - 1) * size + z] * kernel[2][0];
            arr[5] = inimg[elemSize + (y)*size + z] * kernel[2][1];
            arr[8] = inimg[elemSize + (y + 1) * size + z] * kernel[2][2];

            sum = arr[0] + arr[1] + arr[2] + arr[3] + arr[4] + arr[5] + arr[6]
                + arr[7] + arr[8];
            outimg[(y)*size + z] = sum;

        }

        for (y = 1; y < bmpInfoHeader.biHeight - 1; y++) {
            float sum = 0.0;
            arr[2] = arr[1] = inimg[elemSize * (bmpInfoHeader.biWidth - 1) +
                (y - 1) * size + z] * kernel[1][0];

            arr[5] = arr[4] = inimg[elemSize * (bmpInfoHeader.biWidth - 1) +
                y * size + z] * kernel[1][1];

            arr[8] = arr[7] = inimg[elemSize * (bmpInfoHeader.biWidth - 1) +
                (y + 1) * size] * kernel[1][2];

            arr[0] = inimg[elemSize * (bmpInfoHeader.biWidth - 2) +
                (y - 1) * size + z] * kernel[0][0];

            arr[3] = inimg[elemSize * (bmpInfoHeader.biWidth - 2) +
                y * size + z] * kernel[0][1];

            arr[6] = inimg[elemSize * (bmpInfoHeader.biWidth - 2) +
                (y + 1) * size + z] * kernel[0][2];

            sum = arr[0] + arr[1] + arr[2] + arr[3] + arr[4] + arr[5] + arr[6]
                + arr[7] + arr[8];
            outimg[elemSize * (bmpInfoHeader.biWidth - 1) +
                y * size + z] = sum;

        }

        //left-Top
        float sum = 0.0;
        arr[4] = arr[0] = arr[1] = arr[3] = inimg[z] * kernel[1][1];
        arr[5] = arr[2] = inimg[elemSize + z] * kernel[2][1];
        arr[7] = arr[6] = inimg[size + z] * kernel[1][2];
        arr[8] = inimg[elemSize + size + z] * kernel[2][2];

        sum = arr[0] + arr[1] + arr[2] + arr[3] + arr[4] + arr[5] + arr[6]
            + arr[7] + arr[8];
        outimg[z] = sum;


        //left-Bottom
        sum = 0.0;
        arr[4] = arr[3] = arr[6] = arr[7] = inimg[size * (bmpInfoHeader.biHeight - 1) + z]
            * kernel[1][1];
        arr[1] = arr[0] = inimg[size * (bmpInfoHeader.biHeight - 2) + z] * kernel[1][0];
        arr[5] = arr[8] = inimg[elemSize + size * (bmpInfoHeader.biHeight - 1) + z]
            * kernel[2][1];
        arr[2] = inimg[elemSize + size * (bmpInfoHeader.biHeight - 2) + z] * kernel[2][0];

        sum = arr[0] + arr[1] + arr[2] + arr[3] + arr[4] + arr[5] + arr[6]
            + arr[7] + arr[8];
        outimg[size * (bmpInfoHeader.biHeight - 1) + z] = sum;

        //right-Top
        sum = 0.0;
        arr[4] = arr[1] = arr[2] = arr[5] = inimg[elemSize * (bmpInfoHeader.biWidth - 1) + z]
            * kernel[1][1];
        arr[3] = arr[0] = inimg[elemSize * (bmpInfoHeader.biWidth - 2) + z] * kernel[0][1];
        arr[7] = arr[8] = inimg[elemSize * (bmpInfoHeader.biWidth - 1) + size + z] * kernel[1][2];
        arr[6] = inimg[elemSize * (bmpInfoHeader.biWidth - 2) + size + z] * kernel[0][2];
        sum = arr[0] + arr[1] + arr[2] + arr[3] + arr[4] + arr[5] + arr[6]
            + arr[7] + arr[8];
        outimg[elemSize * (bmpInfoHeader.biWidth - 1) + z] = sum;


        //right-Bottom
        sum = 0.0;
        arr[4] = arr[5] = arr[7] = arr[8] = inimg[elemSize * (bmpInfoHeader.biWidth - 1)
            + size * (bmpInfoHeader.biHeight - 1) + z] * kernel[1][1];
        arr[1] = arr[2] = inimg[elemSize * (bmpInfoHeader.biWidth - 1)
            + size * (bmpInfoHeader.biHeight - 2) + z] * kernel[1][0];
        arr[3] = arr[6] = inimg[elemSize * (bmpInfoHeader.biWidth - 2)
            + size * (bmpInfoHeader.biHeight - 1) + z] * kernel[0][1];
        arr[0] = inimg[elemSize * (bmpInfoHeader.biWidth - 2)
            + size * (bmpInfoHeader.biHeight - 2) + z] * kernel[0][0];

        sum = arr[0] + arr[1] + arr[2] + arr[3] + arr[4] + arr[5] + arr[6]
            + arr[7] + arr[8];
        outimg[elemSize * (bmpInfoHeader.biWidth - 1)
            + size * (bmpInfoHeader.biHeight - 1) + z] = sum;

    }

    int main(int argc, char** argv)
    {
        FILE* fp;
        BITMAPFILEHEADER bmpHeader;             /* BMP FILE INFO */
        BITMAPINFOHEADER bmpInfoHeader;     /* BMP IMAGE INFO */
       // RGBQUAD* palrgb;
        ubyte* inimg, * outimg;
        // int x, y, z, imageSize;

        if (argc != 3) {
            fprintf(stderr, "usage : %s input.bmp output.bmp\n", argv[0]);
            return -1;
        }

        /***** read bmp *****/
        if ((fp = fopen(argv[1], "rb")) == NULL) {
            fprintf(stderr, "Error : Failed to open file...₩n");
            return -1;
        }

        /* BITMAPFILEHEADER 구조체의 데이터 */
        fread(&bmpHeader, sizeof(BITMAPFILEHEADER), 1, fp);

        /* BITMAPINFOHEADER 구조체의 데이터 */
        fread(&bmpInfoHeader, sizeof(BITMAPINFOHEADER), 1, fp);

        /* 트루 컬러를 지원하면 변환할 수 없다. */
        if (bmpInfoHeader.biBitCount != 24) {
            perror("This image file doesn't supports 24bit color\n");
            fclose(fp);
            return -1;
        }

        int elemSize = bmpInfoHeader.biBitCount / 8;
        int size = bmpInfoHeader.biWidth * elemSize;
        imageSize = size * bmpInfoHeader.biHeight;
        //outimageSize = (bmpInfoHeader.biWidth+2)*elemSize *
                    //	(bmpInfoHeader.biHeight+2);


        /* 이미지의 해상도(넓이 × 깊이) */
        printf("Resolution : %d x %d\n", bmpInfoHeader.biWidth, bmpInfoHeader.biHeight);
        printf("Bit Count : %d\n", bmpInfoHeader.biBitCount);     /* 픽셀당 비트 수(색상) */
        printf("Image Size : %d\n", imageSize);

        inimg = (ubyte*)malloc(sizeof(ubyte) * imageSize);
        outimg = (ubyte*)malloc(sizeof(ubyte) * imageSize);
        fread(inimg, sizeof(ubyte), imageSize, fp);

        fclose(fp);

        ubyte* d_inimg = NULL, * d_outimg = NULL;
        hipMalloc(&d_inimg, sizeof(ubyte) * imageSize);
        hipMalloc(&d_outimg, sizeof(ubyte) * imageSize);
        hipMemset(d_outimg, 0, sizeof(ubyte) * imageSize);

        //copy host rgb data array to device rgb data array
        hipMemcpy(d_inimg, inimg, sizeof(ubyte) * imageSize, hipMemcpyHostToDevice);

        //define block and grid dimensions
        const dim3 dimGrid((int)ceil((bmpInfoHeader.biWidth / 32)), (int)ceil((bmpInfoHeader.biHeight) / 4));
        const dim3 dimBlock(32, 4);

        //execute cuda kernel
        convertBlur << <dimGrid, dimBlock >> > (d_inimg, d_outimg, bmpInfoHeader.biHeight, bmpInfoHeader.biWidth, elemSize);

        //copy computed blur data array from device to host
        hipMemcpy(outimg, d_outimg, sizeof(ubyte) * imageSize, hipMemcpyDeviceToHost);

        hipFree(d_outimg);
        hipFree(d_inimg);

        /***** write bmp *****/
        if ((fp = fopen(argv[2], "wb")) == NULL) {
            fprintf(stderr, "Error : Failed to open file...₩n");
            return -1;
        }

        /* BITMAPFILEHEADER 구조체의 데이터 */
        fwrite(&bmpHeader, sizeof(BITMAPFILEHEADER), 1, fp);

        /* BITMAPINFOHEADER 구조체의 데이터 */
        fwrite(&bmpInfoHeader, sizeof(BITMAPINFOHEADER), 1, fp);

        //fwrite(inimg, sizeof(ubyte), imageSize, fp);
        fwrite(outimg, sizeof(ubyte), imageSize, fp);

        fclose(fp);

        free(inimg);
        free(outimg);

        return 0;
    }
}
